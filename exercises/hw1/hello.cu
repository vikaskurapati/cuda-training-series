
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

int main(){

  hello<<<3,3>>>();
  cudaCheckError();
  hipDeviceSynchronize();
}

