
#include <hip/hip_runtime.h>
#include <stdio.h>

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    cudaCheckError();
    hipDeviceSynchronize();
    return 0;
}
